#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""


#define FAILURE 0
#define SUCCESS !FAILURE

#define USER_NAME "acr18by"		//replace with your user name

typedef enum MODE { CPU, OPENMP, CUDA, ALL } MODE;
typedef enum OUTPUT_MODE { PPM_BINARY, PPM_PLAIN_TEXT } OUTPUT_MODE;

void print_help();
int process_command_line(int argc, char *argv[]);
unsigned char ** read_data(const char* fname);
unsigned char * gpu_cal(unsigned char *gpu_data);
unsigned char * gpu_cal_optimised(unsigned char *gpu_data);
void cpu_cal();
void openmp_cal();
int output(char * fname);

int c = 0;
unsigned int width = 0;
unsigned int height = 0;
unsigned char ** data;
char *in_file;
char ftype[2];
char *out_file;
int r, g, b;

MODE execution_mode = CPU;
OUTPUT_MODE output_mode = PPM_BINARY;


int main(int argc, char *argv[])
{
	if (process_command_line(argc, argv) == FAILURE)
		return 1;


	//TODO: read input image file (either binary or plain text PPM) 
	printf("Reading data from %s \n", in_file);
	data = read_data(in_file);

	//TODO: execute the mosaic filter based on the mode
	switch (execution_mode) {
	case (CPU): {
		// TODO: starting timing here
		clock_t start = clock(), diff;
		// TODO: calculate the average colour value

		cpu_cal();

		// Output the average colour value for the image
		printf("CPU Average image colour red = %d, green = %d, blue = %d \n", r, g, b);
		// TODO: end timing here
		diff = clock() - start;
		int msec = diff * 1000 / CLOCKS_PER_SEC;
		printf("CPU mode execution time took %d s and %dms\n", msec / 1000, msec % 1000);


		break;
	}
	case (OPENMP): {
		//TODO: starting timing here
		//clock_t start = clock(), diff;
		//TODO: calculate the average colour value
		//double begin, diff;
		//begin = omp_get_wtime();
		openmp_cal();
		// Output the average colour value for the image
		printf("OPENMP Average image colour red = %d, green = %d, blue = %d \n", r, g, b);

		////TODO: end timing here
		//diff = omp_get_wtime() - begin;
		//int msec = diff * 1000;
		//printf("OPENMP mode execution time took %d s and %dms\n", msec / 1000, msec % 1000);
		break;
	}
	case (CUDA): {
		hipEvent_t start, stop;
		float milliseconds = 0;

		unsigned char *gpu_data;
		size_t size = height * width * 3 * sizeof(unsigned char);
		gpu_data = (unsigned char *)malloc(size);
		// transfer data from 2d to 1d
		for (int i = 0, i_1d = 0; i < height; i++) {
			for (int j = 0; j < width * 3; j++, i_1d++) {
				*(gpu_data + i_1d) = *(*(data + i) + j);
			}
		}

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);
		gpu_data = gpu_cal(gpu_data);
		//gpu_data = gpu_cal_optimised(gpu_data);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("CUDA Average image colour red = %d, green = %d, blue = %d \n", r, g, b);
		printf("Execution time is %f ms\n", milliseconds);

		//transfer data from 1d to 2d for output
		for (int i = 0, i_1d = 0; i < height; i++) {
			for (int j = 0; j < width * 3; j++, i_1d++) {
				*(*(data + i) + j) = *(gpu_data + i_1d);
			}
		}
		free(gpu_data);

		output(out_file);
		break;
	}
	case (ALL): {
		//TODO
		clock_t start = clock(), diff;
		hipEvent_t c_start, c_stop;
		float milliseconds = 0;


		// CPU MODE
		cpu_cal();
		printf("\nCPU Average image colour red = %d, green = %d, blue = %d \n", r, g, b);
		diff = clock() - start;
		int msec = diff * 1000 / CLOCKS_PER_SEC;
		printf("CPU mode execution time took %d s and %dms\n", msec / 1000, msec % 1000);
		start = clock();
		
		// OPENMP MODE
		openmp_cal();
		printf("\nOPENMP Average image colour red = %d, green = %d, blue = %d \n", r, g, b);
		diff = clock() - start;
		msec = diff * 1000 / CLOCKS_PER_SEC;
		printf("OPENMP mode execution time took %d s and %dms\n", msec / 1000, msec % 1000);

		// CUDA MODE
		unsigned char *gpu_data;
		size_t size = height * width * 3 * sizeof(unsigned char);
		gpu_data = (unsigned char *)malloc(size);
		// transfer data from 2d to 1d
		for (int i = 0, i_1d = 0; i < height; i++) {
			for (int j = 0; j < width * 3; j++, i_1d++) {
				*(gpu_data + i_1d) = *(*(data + i) + j);
			}
		}
		// CUDA TIME START HERE
		hipEventCreate(&c_start);
		hipEventCreate(&c_stop);
		hipEventRecord(c_start);

		gpu_data = gpu_cal(gpu_data);
		//gpu_data = gpu_cal_optimised(gpu_data);
		hipEventRecord(c_stop);
		hipEventSynchronize(c_stop);
		hipEventElapsedTime(&milliseconds, c_start, c_stop);
		printf("\nCUDA mode execution time took %d s and %dms\n", (int)milliseconds / 1000, (int)milliseconds % 1000);
		printf("CUDA Average image colour red = %d, green = %d, blue = %d \n", r, g, b);

		//transfer data from 1d to 2d for output
		for (int i = 0, i_1d = 0; i < height; i++) {
			for (int j = 0; j < width * 3; j++, i_1d++) {
				*(*(data + i) + j) = *(gpu_data + i_1d);
			}
		}
		free(gpu_data);

		output(out_file);


		break;
	}
	}

	free(data);
	getchar();

	return 0;
}

int process_command_line(int argc, char *argv[]) {
	if (argc < 7) {
		fprintf(stderr, "Error: Missing program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}
	//first argument is always the executable name

	//read in the non optional command line arguments

	c = atoi(argv[1]);

	if (c <= 0) {
		printf("The value of c is invalid.");
		return FAILURE;

	}

	c = pow(2.0, (double)(int)log2(c)); // change the value of c to be valid


	if (!strcmp(argv[2], "CPU")) { execution_mode = CPU; };
	if (!strcmp(argv[2], "OPENMP")) { execution_mode = OPENMP; };
	if (!strcmp(argv[2], "CUDA")) { execution_mode = CUDA; };
	if (!strcmp(argv[2], "ALL")) { execution_mode = ALL; };
	//TODO: read in the input image name
	in_file = argv[4];
	//TODO: read in the output image name
	out_file = argv[6];
	//TODO: read in any optional part 3 arguments
	if (argc > 8) {
		if (!strcmp(argv[8], "PPM_BINARY")) { output_mode = PPM_BINARY; };
		if (!strcmp(argv[8], "PPM_PLAIN_TEXT")) { output_mode = PPM_PLAIN_TEXT; };
	}
			

	

	return SUCCESS;
}

void print_help() {
	printf("mosaic_%s C M -i input_file -o output_file [options]\n", USER_NAME);

	printf("where:\n");
	printf("\tC              Is the mosaic cell size which should be any positive\n"
		"\t               power of 2 number \n");
	printf("\tM              Is the mode with a value of either CPU, OPENMP, CUDA or\n"
		"\t               ALL. The mode specifies which version of the simulation\n"
		"\t               code should execute. ALL should execute each mode in\n"
		"\t               turn.\n");
	printf("\t-i input_file  Specifies an input image file\n");
	printf("\t-o output_file Specifies an output image file which will be used\n"
		"\t               to write the mosaic image\n");
	printf("[options]:\n");
	printf("\t-f ppm_format  PPM image output format either PPM_BINARY (default) or \n"
		"\t               PPM_PLAIN_TEXT\n ");
}

/* Read header information of the file*/
FILE *read_header(FILE *fp) {
	char read_line[10];
	while (1) {
		// exit if reading to the end of file
		if (fgets(read_line, sizeof(read_line), fp) == NULL) {
			return FAILURE;
		}
		// exit if reading to the end line of header
		if (strncmp(read_line, "255", 3) == 0) {
			//size = str_cat(size, input);
			break;
		}
		// file format (either P3 or P6)
		if (strncmp(read_line, "P3", 2) == 0) {
			strcpy(ftype, "P3");
		}
		else if (strncmp(read_line, "P6", 2) == 0) {
			strcpy(ftype, "P6");
		}
		// skip if reading to command line
		else if (strncmp(read_line, "#", 1) == 0) {
			continue;
		}
		// first number is file width and sencond one is height
		else {
			//size = str_cat(size, input);
			// width is not assigned
			if (width == 0) {
				width = atoi(read_line);
			}
			else {
				height = atoi(read_line);
			}
		}
	}

	return fp;
}

/** Read data from the file and do pre-processing
Store the pixel data into the array and return the pointer of the array
*/
unsigned char **read_data(const char *fname) {
	FILE* fp;

	fp = fopen(fname, "rb");
	if (fp == NULL) { perror(fname); return 0; }

	// read header
	fp = read_header(fp);

	if (c > width || c > height) { 
		printf("\nThe value of c is invalide"); 
		
		exit(0); 
	}

	unsigned char **pixel_data = (unsigned char **)malloc(height * sizeof(unsigned char *)); // the memory allocate to store the pixel data

	if (strcmp(ftype, "P3") == 0) {
		for (int row = 0; row < height; row++) {
			pixel_data[row] = (unsigned char *)malloc(width * 3 * sizeof(unsigned char));
		}
		unsigned char *term = (unsigned char *)malloc(sizeof(unsigned char) * 1);
		int i = 0;
		int row, col;
		while (fscanf(fp, "%u", &term) == 1) {
			row = i / (width * 3);
			col = i % (width * 3);
			(*(pixel_data + row))[col] = (unsigned char)term;
			i++;
		}
		fclose(fp);
	}

	if (strcmp(ftype, "P6") == 0) {
		int column, row, k;
		unsigned char * buf = (unsigned char *)malloc(width*height * 3 * sizeof(unsigned char));

		fread(buf, sizeof(unsigned char), width*height * 3, fp); // read all data from the file
		for (row = 0, k = 0; row < height; row++) {
			pixel_data[row] = (unsigned char *)malloc(width * 3 * sizeof(unsigned char));
			for (column = 0; column < width * 3; column++, k++) {
				*(*(pixel_data + row) + column) = (unsigned int)buf[k];
			}
		}
		free(buf);
		fclose(fp);
	}


	return pixel_data;
}

inline double log2(double n) {
	return log(n) / log(2);
}

void cpu_cal() {
	printf("CPU RUNNING\n");
	int i, j, ci, cj; // for index
	int r_ = 0, g_ = 0, b_ = 0; // to calculate the average rgb
	int r_acc = 0, g_acc = 0, b_acc = 0; // accumulated rgb for each block
	int rc = 0, gc = 0, bc = 0; // accumulated rgb for whole image
	int i_c = c, j_c = c; // to solve the boundry overflow problem
	int counter;

	for (i = 0; i < height; i += c) { // row in image
		for (j = 0; j < width * 3; j += 3 * c) { // column in image

			for (ci = i, r_acc = 0, g_acc = 0, b_acc = 0, counter = 0; ci < i + c && ci < height; ci++) {  // row in block
				for (cj = j; cj < j + c * 3 && cj < width * 3; cj += 3, counter++) {  // column in block
					r_acc += *(*(data + ci) + cj + 0);
					g_acc += *(*(data + ci) + cj + 1);
					b_acc += *(*(data + ci) + cj + 2);
				}
			}
			unsigned int
				r_avg = r_acc / counter,
				g_avg = g_acc / counter,
				b_avg = b_acc / counter;

			rc += r_acc;
			gc += g_acc;
			bc += b_acc;

			for (ci = i; ci < i + c && ci < height; ci++) {  // row in block
				for (cj = j; cj < j + c * 3 && cj < width * 3; cj += 3) {  // column in block

					*(*(data + ci) + cj + 0) = r_avg;
					*(*(data + ci) + cj + 1) = g_avg;
					*(*(data + ci) + cj + 2) = b_avg;
				}
			}

			r_ += r_avg;
			g_ += g_avg;
			b_ += b_avg;

		}
	}

	r = rc / (width * height);
	g = gc / (width * height);
	b = bc / (width * height);
}

void openmp_cal() {
	printf("OPENMP RUNNING\n");

	int r_ = 0, g_ = 0, b_ = 0; // to calculate the average rgb
	int rc = 0, gc = 0, bc = 0; // accumulated rgb for whole image
	int i;
	int r_acc, g_acc, b_acc; // accumulated rgb

#pragma omp parallel for reduction(+: r_ , g_ , b_)
	for (i = 0; i < height; i += c) { // row in image
		int j;
		int  ci, cj; // for index
		int counter = 0;
		int r_avg = 0, g_avg = 0, b_avg = 0;

#pragma omp parallel for reduction(+: rc , gc , bc,r_acc, g_acc, b_acc)
		for (j = 0; j < width * 3; j += 3 * c) { // column in image

			for (ci = i, r_acc = 0, g_acc = 0, b_acc = 0, counter = 0; ci < i + c && ci < height; ci++) {  // row in block
				for (cj = j; cj < j + c * 3 && cj < width * 3; cj += 3, counter++) {  // column in block

					r_acc += *(*(data + ci) + cj + 0);
					g_acc += *(*(data + ci) + cj + 1);
					b_acc += *(*(data + ci) + cj + 2);
				}
			}
			r_avg = r_acc / counter;
			g_avg = g_acc / counter;
			b_avg = b_acc / counter;

			rc += r_acc;
			gc += g_acc;
			bc += b_acc;

			for (ci = i; ci < i + c && ci < height; ci++) {  // row in block
				for (cj = j; cj < j + c * 3 && cj < width * 3; cj += 3) {  // column in block
					*(*(data + ci) + cj + 0) = r_avg;
					*(*(data + ci) + cj + 1) = g_avg;
					*(*(data + ci) + cj + 2) = b_avg;
				}
			}
			r_ += r_avg;
			g_ += g_avg;
			b_ += b_avg;
		}
	}
	r = rc / (width * height);
	g = gc / (width * height);
	b = bc / (width * height);

}


/*Pixcel based add up value, per pixcel per thread*/
__global__
void add_up(unsigned char *data, int width, int height, int c, int new_width, int new_height, unsigned int * add_up_data, unsigned int * c_array, unsigned long long int * rgb_all) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < width*height) {
		int loc_row = i / width;
		int loc_col = i % width;
		int loc_row_new = loc_row / c;
		int loc_col_new = loc_col / c;

		atomicAdd((add_up_data + (loc_row_new * new_width + loc_col_new) * 3 + 0), *(data + i * 3 + 0));
		atomicAdd((add_up_data + (loc_row_new * new_width + loc_col_new) * 3 + 1), *(data + i * 3 + 1));
		atomicAdd((add_up_data + (loc_row_new * new_width + loc_col_new) * 3 + 2), *(data + i * 3 + 2));
		atomicAdd((c_array + (loc_row_new * new_width + loc_col_new)), 1); // to count how many pixcel in a mosic block
		atomicAdd((rgb_all + 0), *(data + i * 3 + 0)); // to addup all rgb value
		atomicAdd((rgb_all + 1), *(data + i * 3 + 1)); // to addup all rgb value 
		atomicAdd((rgb_all + 2), *(data + i * 3 + 2)); // to addup all rgb value 
	}
}

/*Mosaic based add up value, per mosic cell per block*/
__global__
void add_up_optimised(unsigned char *data, int width, int height, int c, int new_width, int new_height, unsigned int * add_up_data, unsigned int * c_array, unsigned long long int * rgb_all, int per_mosaic_block_num) {
	__shared__  unsigned int r;
	__shared__  unsigned int g;
	__shared__  unsigned int b;

	int i = (threadIdx.x / c + blockIdx.y*c)*width + (blockIdx.x*c + threadIdx.x % c);
	//					row	* width				 +					col		 	

	int MAXIMUM_WIDTH = c > 32 ? 32 : c;
	int loc_row = i / width;
	int loc_col = i % width;
	int loc_row_new = loc_row / c;
	int loc_col_new = loc_col / c;

	int blockid = blockIdx.x + gridDim.x*blockIdx.y;
	int cellid = blockIdx.x / per_mosaic_block_num + (blockIdx.y / per_mosaic_block_num)*(gridDim.x / per_mosaic_block_num);
	int capacity = c * c;

	if (per_mosaic_block_num > 1) {
		if (blockIdx.x % per_mosaic_block_num == per_mosaic_block_num - 1) {
			if (blockIdx.y % per_mosaic_block_num == per_mosaic_block_num - 1) {
				capacity = c - (per_mosaic_block_num - 1) * 32;
				capacity = capacity * capacity;
			}
			capacity = (c - (per_mosaic_block_num - 1) * 32) * 32;
		}
	}

	if (threadIdx.x < capacity-1) {
		printf("%d %d %d %d %d %d \n", i, cellid, blockid, *(data + i * 3 + 0), *(data + i * 3 + 1), *(data + i * 3 + 2));
		atomicAdd(&r, *(data + i * 3 + 0));
		atomicAdd(&g, *(data + i * 3 + 1));
		atomicAdd(&b, *(data + i * 3 + 2));	
	}	

	__syncthreads();

	if (threadIdx.x == 0) {
		printf("---%d %d %d %d %d %d \n", i, cellid, blockid, r, g, b);
		atomicAdd((add_up_data + cellid * 3 + 0), r);
		atomicAdd((add_up_data + cellid * 3 + 1), g);
		atomicAdd((add_up_data + cellid * 3 + 2), b);
		atomicAdd((c_array + cellid), capacity); // to count how many pixcel in a mosic block

		atomicAdd((rgb_all + 0), r); // to addup all rgb value
		atomicAdd((rgb_all + 1), g); // to addup all rgb value 
		atomicAdd((rgb_all + 2), b); // to addup all rgb value 
	}
}

/*calculate the average value in mosaic cell and replace the original value by the value in mosaic cell*/
__global__
void avg(unsigned char * data, int width, int height, int c, int new_width, int new_height, unsigned int * add_up_data, unsigned int * c_array) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < width*height) {
		int loc_row = i / width;
		int loc_col = i % width;
		int loc_row_new = loc_row / c;
		int loc_col_new = loc_col / c;

		*(data + i * 3 + 0) = (*(add_up_data + (loc_row_new * new_width + loc_col_new) * 3 + 0) / *(c_array + (loc_row_new * new_width + loc_col_new)));
		*(data + i * 3 + 1) = (*(add_up_data + (loc_row_new * new_width + loc_col_new) * 3 + 1) / *(c_array + (loc_row_new * new_width + loc_col_new)));
		*(data + i * 3 + 2) = (*(add_up_data + (loc_row_new * new_width + loc_col_new) * 3 + 2) / *(c_array + (loc_row_new * new_width + loc_col_new)));
	}
}

unsigned char * gpu_cal(unsigned char *gpu_data) {
	size_t size = height * width * 3 * sizeof(unsigned char);
	int add_up_data_width;
	int add_up_data_height;
	unsigned int *add_up_data_dev, *add_up_data_host; // to calculate the total rgb value in a mosic cell 
	unsigned int *c_array_dev, *c_array_host; // to count how many pixels in a mosic cell

	unsigned char *data_1d_dev; // image data
	unsigned long long int *rgb_all_dev, *rgb_all_host; // all rgb value addup
	const int BLOCK_SIZE = 512;

	add_up_data_width = width % c == 0 ? width / c : (width / c + 1);
	add_up_data_height = height % c == 0 ? height / c : (height / c + 1);

	add_up_data_host = (unsigned int *)malloc(add_up_data_width * add_up_data_height * 3 * sizeof(unsigned int));
	c_array_host = (unsigned int *)malloc(add_up_data_width * add_up_data_height * sizeof(unsigned int));
	rgb_all_host = (unsigned long long int *)malloc(3 * sizeof(unsigned long long int));

	hipMalloc(&data_1d_dev, size);
	hipMalloc(&add_up_data_dev, add_up_data_width * add_up_data_height * 3 * sizeof(unsigned int));
	hipMalloc(&c_array_dev, add_up_data_width * add_up_data_height * sizeof(int));
	hipMalloc(&rgb_all_dev, 3 * sizeof(unsigned long long int));
	hipMemcpy(data_1d_dev, gpu_data, size, hipMemcpyHostToDevice);


	// excutive addup kernel function
	add_up << < ((size / 3) / BLOCK_SIZE) > 0 ? (size / 3) / BLOCK_SIZE : 1, BLOCK_SIZE >> > (data_1d_dev, width, height, c, add_up_data_width, add_up_data_height, add_up_data_dev, c_array_dev, rgb_all_dev);
	
	// excutive average kernel function
	avg << < ((size / 3) / BLOCK_SIZE) > 0 ? (size / 3) / BLOCK_SIZE : 1, BLOCK_SIZE >> > (data_1d_dev, width, height, c, add_up_data_width, add_up_data_height, add_up_data_dev, c_array_dev);

	hipMemcpy(rgb_all_host, rgb_all_dev, 3 * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_data, data_1d_dev, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	r = *(rgb_all_host + 0) / width / height;
	g = *(rgb_all_host + 1) / width / height;
	b = *(rgb_all_host + 2) / width / height;

	hipFree(add_up_data_dev);
	free(add_up_data_host);

	hipFree(c_array_dev);
	free(c_array_host);

	hipFree(data_1d_dev);

	hipFree(rgb_all_dev);
	free(rgb_all_host);

	return gpu_data;
}

unsigned char * gpu_cal_optimised(unsigned char *gpu_data) {
	unsigned int *add_up_data_dev, *add_up_data_host; // to calculate the total rgb value in a mosic cell 
	unsigned int *c_array_dev, *c_array_host; // to count how many pixels in a mosic cell
	unsigned char *data_1d_dev; // image data
	unsigned long long int *rgb_all_dev, *rgb_all_host; // all rgb value addup
	const int BLOCK_SIZE = 512;

	int add_up_data_width = width % c == 0 ? width / c : (width / c + 1);
	int add_up_data_height = height % c == 0 ? height / c : (height / c + 1);
	int BLOCKBIM = c > 32 ? 32 : c; // maximun is 32
	int BLOCK_PER_MOSAIC = c / ( BLOCKBIM + 1 ) + 1;
	
	dim3 block(BLOCKBIM*BLOCKBIM, 1, 1);
	dim3 grid(BLOCK_PER_MOSAIC * add_up_data_width, BLOCK_PER_MOSAIC * add_up_data_height, 1);
	size_t size = height * width * 3 * sizeof(unsigned char);

	add_up_data_host = (unsigned int *)malloc(add_up_data_width * add_up_data_height * 3 * sizeof(unsigned int));
	c_array_host = (unsigned int *)malloc(add_up_data_width * add_up_data_height * sizeof(unsigned int));
	rgb_all_host = (unsigned long long int *)malloc(3 * sizeof(unsigned long long int));

	hipMalloc(&data_1d_dev, size);
	hipMalloc(&add_up_data_dev, add_up_data_width * add_up_data_height * 3 * sizeof(unsigned int));
	hipMalloc(&c_array_dev, add_up_data_width * add_up_data_height * sizeof(int));
	hipMalloc(&rgb_all_dev, 3 * sizeof(unsigned long long int));
	hipMemcpy(data_1d_dev, gpu_data, size, hipMemcpyHostToDevice);

	add_up_optimised << < grid, block >> > (data_1d_dev, width, height, c, add_up_data_width, add_up_data_height, add_up_data_dev, c_array_dev, rgb_all_dev, BLOCK_PER_MOSAIC);
	
	avg << < ((size / 3) / BLOCK_SIZE) > 0 ? (size / 3) / BLOCK_SIZE : 1, BLOCK_SIZE >> > (data_1d_dev, width, height, c, add_up_data_width, add_up_data_height, add_up_data_dev, c_array_dev);

	hipDeviceSynchronize();

	hipMemcpy(rgb_all_host, rgb_all_dev, 3 * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipMemcpy(c_array_host, c_array_dev, add_up_data_width * add_up_data_height * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_data, data_1d_dev, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	r = *(rgb_all_host + 0) / width / height;
	g = *(rgb_all_host + 1) / width / height;
	b = *(rgb_all_host + 2) / width / height;

	hipFree(add_up_data_dev);
	free(add_up_data_host);

	hipFree(c_array_dev);
	free(c_array_host);

	hipFree(data_1d_dev);

	hipFree(rgb_all_dev);
	free(rgb_all_host);

	return gpu_data;
}

int output(char * fname) {
	FILE* fp;
	int row, column, p_i, index, i;
	char* all_data;
	unsigned char* bin_data;
	char str_buf[10];
	char* char_num = (char*)malloc(4);
	int s = 0;

	printf("\nStart writing---------------\n");

	switch (output_mode) {

	case(PPM_PLAIN_TEXT):

		fp = fopen(fname, "w");
		fputs("P3\n", fp);
		fputs("# COM6521 Assignment test output\n", fp);
		sprintf(str_buf, "%d\n", width);
		fputs(str_buf, fp);
		sprintf(str_buf, "%d\n", height);
		fputs(str_buf, fp);
		sprintf(str_buf, "%d\n", 255);
		fputs(str_buf, fp);

		// format all data into string and write it into file 
		all_data = (char *)malloc(width*height * 13 * sizeof(char));
		memset(all_data, '\0', width*height * 13 * sizeof(char));

		for (row = 0, p_i = 0, index = 0; row < height; row++, p_i++, index++) {
			for (column = 0; column < width * 3; column++, i = 0, index++) {   // process number by number
				sprintf(char_num, "%d\0", *(*(data + row) + column));
				for (i = 0; *(char_num + i) != '\0' && i < 3; i++, index++) {
					*(all_data + index) = *(char_num + i);
				}
				if (p_i == 3) {
					*(all_data + index) = '\t';
					p_i = 0;
				}
				else {
					*(all_data + index) = ' ';
				}
			}
			*(all_data + index) = '\n';
		}
		fputs(all_data, fp);
		free(all_data);

		fclose(fp);
		break;


	case(PPM_BINARY):
		fp = fopen(fname, "wb");
		fputs("P6\n", fp);
		fputs("# COM6521 Assignment test output\n", fp);
		sprintf(str_buf, "%d\n", width);
		fputs(str_buf, fp);
		sprintf(str_buf, "%d\n", height);
		fputs(str_buf, fp);
		sprintf(str_buf, "%d\n", 255);
		fputs(str_buf, fp);
		bin_data = (unsigned char*)malloc(width*height * 3 * sizeof(unsigned char));

		for (row = 0, index = 0; row < height; row++) {
			for (column = 0; column < width * 3; column++, index++) {
				*(bin_data + index) = (unsigned char)*(*(data + row) + column);
			}
		}

		fwrite(bin_data, sizeof(unsigned char), width*height * 3 * sizeof(unsigned char), fp);
		fclose(fp);
		free(bin_data);

		break;
	}
	printf("The file has been saved as %s", out_file);
	return SUCCESS;
}


